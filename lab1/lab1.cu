﻿#include "hip/hip_runtime.h"

#include <iostream>
#include <math.h>
#include <cstdlib>

using namespace std;

__global__ void kernel(double b, double q, double* ans, long long n) { //отличное от C++ (__global__)
	long long i, idx = blockDim.x * blockIdx.x + threadIdx.x;			// Абсолютный номер потока 
	long long offset = blockDim.x * gridDim.x;						// Общее кол-во потоков
	for (i = idx; i < n; i += offset) // Для всех требование - внутри цикла for()
		ans[i] = b*pow(q, i);
}

int main() {
	ios::sync_with_stdio(false);
	cin.tie(nullptr);

	long long n;
	double b = 1;
	double q = 3;
	cin >> n;
	double* answer = (double*)malloc(sizeof(double) * n); //выделение массива ответа

	double* result;
	hipMalloc(&result, sizeof(double) * n); //выделение массива на устройстве 
	hipMemcpy(result, answer, sizeof(double) * n, hipMemcpyHostToDevice);

	kernel <<<256, 256>>> (b, q, result, n); //отличное от C++ (<<<>>>), стандартная функция
	// Многопоточное
	// 256 блоков и 256 потоков(Thread)


	hipMemcpy(answer, result, sizeof(double) * n, hipMemcpyDeviceToHost);
	hipFree(result);

	cout.precision(10);
	cout.setf(ios::scientific);
	for (long long i = 0; i < n; i++)
		cout << answer[i] << ' ';
	cout << endl;
	free(answer);
	cin >> n;
	return 0;
}
